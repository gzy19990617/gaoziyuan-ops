
#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>
#include <chrono>


#define NUM 512

//v4:使用寄存器，两级缓存

// 提取k维度，从内积变为外积，其实就相当于对shread_memrory做分块
// 先拿一列，再拿一行，算出来一块
// 相当于从shared_memory中取的次数变少了

# define FETCH_FLOA4(pointer)(reinterpret_cast<float4 *>(&(pointer))[0])

template<unsigned int M_NUM_PER_BLOCK, unsigned int N_NUM_PER_BLOCK, unsigned int K_NUM_PER_BLOCK, unsigned int NUM_PER_THREAD>
__global__ void matrixMulCUDA(float* C,  float* A,  float* B, int M, int N, int K) {
    
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // reshape索引，便于寄存器取数据
    int tid = ty * blockDim.x + tx;
    int ctx = tid % 16;
    int cty = tid / 16;

    __shared__ float a_shared[M_NUM_PER_BLOCK][K_NUM_PER_BLOCK];
    __shared__ float b_shared[K_NUM_PER_BLOCK][N_NUM_PER_BLOCK];

    // 申请寄存器
    constexpr int REG_NUM = NUM_PER_THREAD / 2;
    float a_reg[REG_NUM] = {0.f};
    float b_reg[REG_NUM] = {0.f};

    float temp[REG_NUM][REG_NUM] = {0.f};

    int x = tx * NUM_PER_THREAD +  blockIdx.x * M_NUM_PER_BLOCK;
    int y = ty +  blockIdx.y * N_NUM_PER_BLOCK;
    
    // K方向做步进
    for (int s = 0; s < K; s += K_NUM_PER_BLOCK) {
        //可以根据最local版本方案去写索引
        // a_shared[threadIdx.y][threadIdx.x] = A[y * K + threadIdx.x + s];
        // b_shared[threadIdx.y][threadIdx.x] = B[(s + threadIdx.y) * N + x];
        
        FETCH_FLOA4(a_shared[ty][tx * NUM_PER_THREAD]) = FETCH_FLOA4(A[y * K + tx * NUM_PER_THREAD + s]);
        FETCH_FLOA4(b_shared[ty][tx * NUM_PER_THREAD]) = FETCH_FLOA4(B[N * (s + ty) + x]);
        __syncthreads();

        for (int k = 0; k < K_NUM_PER_BLOCK; k++) {
            a_reg[0] = a_shared[cty * 2][k];
            a_reg[1] = a_shared[cty * 2 + 1][k];
            b_reg[0] = b_shared[k][ctx * 2];
            b_reg[1] = b_shared[k][ctx * 2 + 1];
            for (int i = 0; i < REG_NUM; i++) {
                for (int j = 0; j < REG_NUM; j++) {
                    temp[i][j] += a_reg[i] * b_reg[j];
                }
            }
        }
        __syncthreads();
    }


    // 因为改变了线程排布，上面的全局索引用不了
    float * C_ptr_start = C + blockIdx.x * N_NUM_PER_BLOCK + blockIdx.y * M_NUM_PER_BLOCK * N;
    for (int i = 0; i < REG_NUM; i++) {
            for (int j = 0; j < REG_NUM; j++) {
                C_ptr_start[N *(cty * 2 + i) + ctx * 2 + j] = temp[i][j];
            }
        }
}

// CPU 实现：矩阵乘法
void matrixMulCPU(float* C, const float* A, const float* B, int M, int N, int K) {
    for (int row = 0; row < M; ++row) {
        for (int col = 0; col < N; ++col) {
            float value = 0.0f;
            for (int i = 0; i < K; ++i) {
                value += A[row * K + i] * B[i * N + col]; // 点积计算
            }
            C[row * N + col] = value; // 写入结果矩阵
        }
    }
}

// 比较两个矩阵的精度
bool compareMatrices(const float* C1, const float* C2, int M, int N, float epsilon = 1e-2) {
    for (int i = 0; i < M * N; ++i) {
        if (fabs(C1[i] - C2[i]) > epsilon) {
            std::cout << "Mismatch at index " << i << ": CUDA=" << C1[i] << ", CPU=" << C2[i] << std::endl;
            return false;
        }
    }
    return true;
}

int main() {
    int M = NUM; // A 的行数
    int K = NUM; // A 的列数，B 的行数
    int N = NUM; // B 的列数

    // 分配主机内存
    float* h_A = new float[M * K];
    float* h_B = new float[K * N];
    float* h_C_cpu = new float[M * N];
    float* h_C_cuda = new float[M * N];

    // 初始化矩阵 A 和 B
    for (int i = 0; i < M * K; ++i) h_A[i] = static_cast<float>(rand()) / RAND_MAX;
    for (int i = 0; i < K * N; ++i) h_B[i] = static_cast<float>(rand()) / RAND_MAX;

    // 分配设备内存
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, M * K * sizeof(float));
    hipMalloc(&d_B, K * N * sizeof(float));
    hipMalloc(&d_C, M * N * sizeof(float));

    // 将数据从主机复制到设备
    hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice);


    constexpr int M_NUM_PER_BLOCK = 32;
    constexpr int N_NUM_PER_BLOCK = 32;
    constexpr int K_NUM_PER_BLOCK = 32;
    constexpr int NUM_PER_THREAD = 4;

    // 定义 CUDA 线程块和网格大小
    // 这里使用了STRIDE来控制网格大小，以减少总的线程数。这样可以减小总的线程数，从而降低资源消耗和可能的内存溢出风险。
    dim3 blockSize(8, 32); // 每个线程块 16x16 线程
    dim3 gridSize(M / M_NUM_PER_BLOCK, N / N_NUM_PER_BLOCK);

    // 执行 CUDA 核函数
    auto start_cuda = std::chrono::high_resolution_clock::now();
    matrixMulCUDA<M_NUM_PER_BLOCK, N_NUM_PER_BLOCK, K_NUM_PER_BLOCK, NUM_PER_THREAD><<<gridSize, blockSize>>>(d_C, d_A, d_B, M, N, K);
    hipDeviceSynchronize(); // 等待 GPU 完成
    auto end_cuda = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_cuda = end_cuda - start_cuda;

    // 将结果从设备复制回主机
    hipMemcpy(h_C_cuda, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    // 执行 CPU 实现
    auto start_cpu = std::chrono::high_resolution_clock::now();
    matrixMulCPU(h_C_cpu, h_A, h_B, M, N, K);
    auto end_cpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_cpu = end_cpu - start_cpu;

    // 比较 GPU 和 CPU 的结果
    if (compareMatrices(h_C_cuda, h_C_cpu, M, N)) {
        std::cout << "CUDA and CPU results match!" << std::endl;
    } else {
        std::cout << "CUDA and CPU results do not match!" << std::endl;
    }

    // 输出运行时间
    std::cout << "CUDA time: " << elapsed_cuda.count() << " seconds" << std::endl;
    std::cout << "CPU time: " << elapsed_cpu.count() << " seconds" << std::endl;

    // 释放内存
    delete[] h_A;
    delete[] h_B;
    delete[] h_C_cpu;
    delete[] h_C_cuda;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}