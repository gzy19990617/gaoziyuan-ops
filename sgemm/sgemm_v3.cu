
#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>
#include <chrono>


#define NUM 512

//v3:使用float4取4个连续的元素

// 只优化从globol到shared的搬运使用float4

# define FETCH_FLOA4(pointer)(reinterpret_cast<float4 *>(&(pointer))[0])

template<unsigned int M_NUM_PER_BLOCK, unsigned int N_NUM_PER_BLOCK, unsigned int K_NUM_PER_BLOCK, unsigned int NUM_PER_THREAD>
__global__ void matrixMulCUDA(float* C,  float* A,  float* B, int M, int N, int K) {
    
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    __shared__ float a_shared[M_NUM_PER_BLOCK][K_NUM_PER_BLOCK];
    __shared__ float b_shared[K_NUM_PER_BLOCK][N_NUM_PER_BLOCK];


    int x = tx * NUM_PER_THREAD +  blockIdx.x * M_NUM_PER_BLOCK;
    int y = ty +  blockIdx.y * N_NUM_PER_BLOCK;

    // 一个线程处理4个数,一维的,是一条
    float value[NUM_PER_THREAD] = {0.f};
    
    // K方向做步进
    for (int s = 0; s < K; s += K_NUM_PER_BLOCK) {
        //可以根据最local版本方案去写索引
        // a_shared[threadIdx.y][threadIdx.x] = A[y * K + threadIdx.x + s];
        // b_shared[threadIdx.y][threadIdx.x] = B[(s + threadIdx.y) * N + x];
        
        FETCH_FLOA4(a_shared[ty][tx * NUM_PER_THREAD]) = FETCH_FLOA4(A[y * K + tx * NUM_PER_THREAD + s]);
        FETCH_FLOA4(b_shared[ty][tx * NUM_PER_THREAD]) = FETCH_FLOA4(B[N * (s + ty) + x]);
        // a_shared[ty][tx * M_NUM_PER_BLOCK] = A[y * K + tx * M_NUM_PER_BLOCK + s];
        // a_shared[ty][tx * M_NUM_PER_BLOCK + 1] = A[y * K + tx * M_NUM_PER_BLOCK + 1 + s];
        // a_shared[ty][tx * M_NUM_PER_BLOCK + 2] = A[y * K + tx * M_NUM_PER_BLOCK + 2 + s];
        // a_shared[ty][tx * M_NUM_PER_BLOCK + 3] = A[y * K + tx * M_NUM_PER_BLOCK + 3 + s];

        // b_shared[ty][tx * M_NUM_PER_BLOCK] = B[N * (s + threadIdx.y) + x];
        // b_shared[ty][tx * M_NUM_PER_BLOCK + 1] = B[N * (s + threadIdx.y) + x + 1];
        // b_shared[ty][tx * M_NUM_PER_BLOCK + 2] = B[N * (s + threadIdx.y) + x + 2];
        // b_shared[ty][tx * M_NUM_PER_BLOCK + 3] = B[N * (s + threadIdx.y) + x + 3];
        __syncthreads();

        for (int i = 0; i < NUM_PER_THREAD; i++)  {
                for (int k = 0; k < K_NUM_PER_BLOCK; k++) { // K 维度是step
                    value[i] += a_shared[ty][k] * b_shared[k][tx * NUM_PER_THREAD + i];
                }
            }

        __syncthreads();
    }
     for (int i = 0; i < NUM_PER_THREAD; i++)  {
            C[y * N  + x + i] += value[i];
    }
}

// CPU 实现：矩阵乘法
void matrixMulCPU(float* C, const float* A, const float* B, int M, int N, int K) {
    for (int row = 0; row < M; ++row) {
        for (int col = 0; col < N; ++col) {
            float value = 0.0f;
            for (int i = 0; i < K; ++i) {
                value += A[row * K + i] * B[i * N + col]; // 点积计算
            }
            C[row * N + col] = value; // 写入结果矩阵
        }
    }
}

// 比较两个矩阵的精度
bool compareMatrices(const float* C1, const float* C2, int M, int N, float epsilon = 1e-2) {
    for (int i = 0; i < M * N; ++i) {
        if (fabs(C1[i] - C2[i]) > epsilon) {
            std::cout << "Mismatch at index " << i << ": CUDA=" << C1[i] << ", CPU=" << C2[i] << std::endl;
            return false;
        }
    }
    return true;
}

int main() {
    int M = NUM; // A 的行数
    int K = NUM; // A 的列数，B 的行数
    int N = NUM; // B 的列数

    // 分配主机内存
    float* h_A = new float[M * K];
    float* h_B = new float[K * N];
    float* h_C_cpu = new float[M * N];
    float* h_C_cuda = new float[M * N];

    // 初始化矩阵 A 和 B
    for (int i = 0; i < M * K; ++i) h_A[i] = static_cast<float>(rand()) / RAND_MAX;
    for (int i = 0; i < K * N; ++i) h_B[i] = static_cast<float>(rand()) / RAND_MAX;

    // 分配设备内存
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, M * K * sizeof(float));
    hipMalloc(&d_B, K * N * sizeof(float));
    hipMalloc(&d_C, M * N * sizeof(float));

    // 将数据从主机复制到设备
    hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice);


    constexpr int M_NUM_PER_BLOCK = 32;
    constexpr int N_NUM_PER_BLOCK = 32;
    constexpr int K_NUM_PER_BLOCK = 32;
    constexpr int NUM_PER_THREAD = 4;

    // 定义 CUDA 线程块和网格大小
    // 这里使用了STRIDE来控制网格大小，以减少总的线程数。这样可以减小总的线程数，从而降低资源消耗和可能的内存溢出风险。
    dim3 blockSize(8, 32); // 每个线程块 16x16 线程
    dim3 gridSize(M / M_NUM_PER_BLOCK, N / N_NUM_PER_BLOCK);

    // 执行 CUDA 核函数
    auto start_cuda = std::chrono::high_resolution_clock::now();
    matrixMulCUDA<M_NUM_PER_BLOCK, N_NUM_PER_BLOCK, K_NUM_PER_BLOCK, NUM_PER_THREAD><<<gridSize, blockSize>>>(d_C, d_A, d_B, M, N, K);
    hipDeviceSynchronize(); // 等待 GPU 完成
    auto end_cuda = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_cuda = end_cuda - start_cuda;

    // 将结果从设备复制回主机
    hipMemcpy(h_C_cuda, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    // 执行 CPU 实现
    auto start_cpu = std::chrono::high_resolution_clock::now();
    matrixMulCPU(h_C_cpu, h_A, h_B, M, N, K);
    auto end_cpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_cpu = end_cpu - start_cpu;

    // 比较 GPU 和 CPU 的结果
    if (compareMatrices(h_C_cuda, h_C_cpu, M, N)) {
        std::cout << "CUDA and CPU results match!" << std::endl;
    } else {
        std::cout << "CUDA and CPU results do not match!" << std::endl;
    }

    // 输出运行时间
    std::cout << "CUDA time: " << elapsed_cuda.count() << " seconds" << std::endl;
    std::cout << "CPU time: " << elapsed_cpu.count() << " seconds" << std::endl;

    // 释放内存
    delete[] h_A;
    delete[] h_B;
    delete[] h_C_cpu;
    delete[] h_C_cuda;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}