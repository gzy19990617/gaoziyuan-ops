#include <stdio.h>
#include <hip/hip_runtime.h>





typedef float FLOAT;


__global__ void vec_add(FLOAT* x, FLOAT* y, FLOAT* z, int N) {

    // 1D
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        z[idx] = y[idx] + x[idx];
    }
}

void vec_add_cpu(FLOAT* x, FLOAT* y, FLOAT* z, int N) {
    for (int i =0; i < N; i++) {
        z[i] = y[i] + x[i];
    }
}

int main() {
    int N = 10000;
    int nbytes = N * sizeof(FLOAT);


    // 一维grid，每个block里面有256个线程
    int bs = 256;
    int s = ceil((N + bs - 1.0) / bs);
    dim3 grid(s);


    FLOAT *dx, *hx;
    FLOAT *dy, *hy;
    FLOAT *dz, *hz;

    hipMalloc((void **)&dx, nbytes);
    hipMalloc((void **)&dy, nbytes);
    hipMalloc((void **)&dz, nbytes);

    hx = (FLOAT*) malloc(nbytes);
    hy = (FLOAT*) malloc(nbytes);
    hz = (FLOAT*) malloc(nbytes);

    for (int i =0;i<N;i++){
        hx[i] = 1;
        hy[i] = 1;
    }

    hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(dy, hy, nbytes, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    float milliseconds = 0;

    vec_add<<<grid, bs>>>(dx,dy,dz,N);

     hipEventRecord(stop);
     hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);  

    hipMemcpy(hz, dz, nbytes, hipMemcpyDeviceToHost);

     /* CPU compute */
    FLOAT* hz_cpu_res = (FLOAT *) malloc(nbytes);
    vec_add_cpu(hx, hy, hz_cpu_res, N);

    for (int i = 0; i < N; ++i) {
        if (fabs(hz_cpu_res[i] - hz[i]) > 1e-6) {
            printf("Result verification failed at element index %d!\n", i);
        }
    }
    printf("Result right\n");
    hipFree(dx);
    hipFree(dy);
    hipFree(dz);

    free(hx);
    free(hy);
    free(hz);
    free(hz_cpu_res);

    return 0;
}