#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


#include <time.h>
#include <sys/time.h>

#define THREAD_PER_BLOCK 256

// v2:避免线程分化，一个warp是32个线程，同一个warp中的不同分支失去了并发行。
// 不同warp之间又保留了并发性，会存在不一致，容易导致死锁

// a = (cond ? x[i]:0.f) 这种三元表达符号不会导致分支
// 让每轮迭代的前一半线程负责运算，组成一个warp

__global__ void reduce1(float* d_a, float* d_out) {
    __shared__ float s_a[THREAD_PER_BLOCK];

    // 搬运数据到共享内存中，每个线程搬运一个元素
    float* input_begein = d_a + blockIdx.x * blockDim.x;
    s_a[threadIdx.x] = input_begein[threadIdx.x];
    // 搬运完需要进行同步
    __syncthreads();

    for (int i = 1; i < blockDim.x; i *= 2) {
        if (threadIdx.x < blockDim.x / (2 * i)) {
            // 第一轮：0号线程负责，0+1. 1号线程负责，1+2. 以此类推
            // 第二轮：0号线程负责，0+2. 1号线程负责，4+6. 以此类推
            int index = threadIdx.x * (2 * i);
            s_a[index] += s_a[index + i];
        }
        // 每一次要等这一轮计算完
        __syncthreads();
    }
    //最终每个block把计算结果放在第一个索引的位置
    if (threadIdx.x == 0) {
        d_out[blockIdx.x] = input_begein[0];
    }
}

// __global__ void reduce1(float* d_a, float* d_out) {
//     __shared__ float s_a[THREAD_PER_BLOCK];
    
//     int tid = threadIdx.x;
//     int global_tid = blockIdx.x * blockDim.x + tid;

//     // 搬运数据到共享内存中，每个线程搬运一个元素
//     s_a[tid] = d_a[global_tid];

//     // 搬运完需要进行同步
//     __syncthreads();

//     for (int i = 1; i < blockDim.x; i *= 2) {
//         if (tid % (2 * i) == 0) {
//             d_a[global_tid] += d_a[global_tid + i];
//         }
//         // 每一次要等这一轮计算完
//         __syncthreads();
//     }
//     //最终每个block把计算结果放在第一个索引的位置
//     if (tid == 0) {
//         d_out[blockIdx.x] = d_a[global_tid];
//     }
// }


bool check(float *out,float *res,int n){
    for(int i=0;i<n;i++){
        if (abs(out[i] - res[i]) > 0.005) {
            return false;
        }
    }
    return true;
}

int main() {
    printf("hello \n");

    const int N=32*1024*1024;
    float *a=(float *)malloc(N*sizeof(float));
    float *d_a;
    hipMalloc((void **)&d_a,N*sizeof(float));


    int block_num = N / THREAD_PER_BLOCK;

    float *out = (float *)malloc((block_num * sizeof(float)));
    float *d_out;
    hipMalloc((void **)&d_out, block_num * sizeof(float));

    float *res=(float *)malloc((N/THREAD_PER_BLOCK)*sizeof(float));

    for(int i=0;i<N;i++){
        a[i]=1.2;
    }

    for(int i=0;i<block_num;i++){
        float cur=0;
        for(int j=0;j<THREAD_PER_BLOCK;j++){
            cur+=a[i*THREAD_PER_BLOCK+j];
        }
        res[i]=cur;
    }

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(block_num);
    dim3 Block(THREAD_PER_BLOCK);
    

    reduce1<<<Grid, Block>>>(d_a, d_out);

    hipMemcpy(out, d_out, block_num*sizeof(float),hipMemcpyDeviceToHost);

     if(check(out,res,block_num))printf("the ans is right\n");
    else{
        printf("the ans is wrong\n");
        for(int i=0;i<block_num;i++){
            printf("%lf ",out[i]);
        }
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_out);

    return 0;
}