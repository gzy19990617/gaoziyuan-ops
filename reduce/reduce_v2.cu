#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


#include <time.h>
#include <sys/time.h>

#define THREAD_PER_BLOCK 256

// v2:避免线程分化，一个warp是32个线程，同一个warp中的不同分支失去了并发行。
// 0.638784 ms
// 不同warp之间又保留了并发性，会存在不一致，容易导致死锁

// a = (cond ? x[i]:0.f) 这种三元表达符号不会导致分支
// 让每轮迭代的前一半线程负责运算，组成一个warp

__global__ void reduce1(float* d_a, float* d_out) {
    __shared__ float s_a[THREAD_PER_BLOCK];

    // 搬运数据到共享内存中，每个线程搬运一个元素
    float* input_begein = d_a + blockIdx.x * blockDim.x;
    s_a[threadIdx.x] = input_begein[threadIdx.x];
    // 搬运完需要进行同步
    __syncthreads();

    for (int i = 1; i < blockDim.x; i *= 2) {
        if (threadIdx.x < blockDim.x / (2 * i)) {
            // 前一半的线程都做运算，后一半的线程什么都不做
            // 0号线程处理第一0个第一个元素；1号线程处理第2个第3；2处理4、5；之前是0号处理0、1，2号处理2、3元素
            int index = threadIdx.x * (2 * i);
            s_a[index] += s_a[index + i];
        }
        // 每一次要等这一轮计算完
        __syncthreads();
    }
    //最终每个block把计算结果放在第一个索引的位置
    if (threadIdx.x == 0) {
        d_out[blockIdx.x] = s_a[0];
    }
}

bool check(float *out,float *res,int n){
    for(int i=0;i<n;i++){
        if (abs(out[i] - res[i]) > 0.005) {
            return false;
        }
    }
    return true;
}

int main() {
    float milliseconds = 0;
    printf("hello \n");

    const int N=32*1024*1024;
    float *a=(float *)malloc(N*sizeof(float));
    float *d_a;
    hipMalloc((void **)&d_a,N*sizeof(float));


    int block_num = N / THREAD_PER_BLOCK;

    float *out = (float *)malloc((block_num * sizeof(float)));
    float *d_out;
    hipMalloc((void **)&d_out, block_num * sizeof(float));

    float *res=(float *)malloc((N/THREAD_PER_BLOCK)*sizeof(float));

    for(int i=0;i<N;i++){
        a[i]=1.2;
    }

    for(int i=0;i<block_num;i++){
        float cur=0;
        for(int j=0;j<THREAD_PER_BLOCK;j++){
            cur+=a[i*THREAD_PER_BLOCK+j];
        }
        res[i]=cur;
    }

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(block_num);
    dim3 Block(THREAD_PER_BLOCK);
    

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    reduce1<<<Grid, Block>>>(d_a, d_out);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(out, d_out, block_num*sizeof(float),hipMemcpyDeviceToHost);

     if(check(out,res,block_num))printf("the ans is right\n");
    else{
        printf("the ans is wrong\n");
        for(int i=0;i<block_num;i++){
            printf("%lf ",out[i]);
        }
        printf("\n");
    }
    printf("reduce_v0 latency = %f ms\n", milliseconds);

    hipFree(d_a);
    hipFree(d_out);

    return 0;
}