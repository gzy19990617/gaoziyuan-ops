#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


#include <time.h>
#include <sys/time.h>

#define THREAD_PER_BLOCK 256

// v4: 让每个线程做的事情更多
// 0.236000 ms
// planA:减少block数量；保持block中thread的数量；让每个thread处理更多的数据

__global__ void reduce1(float* d_a, float* d_out) {
    __shared__ float s_a[THREAD_PER_BLOCK];

    float* input_begein = d_a + blockIdx.x * blockDim.x * 2;
    s_a[threadIdx.x] = input_begein[threadIdx.x] + input_begein[threadIdx.x + blockDim.x];
    // 搬运完需要进行同步
    __syncthreads();

    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (threadIdx.x < i) {
            s_a[threadIdx.x] += s_a[threadIdx.x + i];
            __syncthreads();
        }
    }
    if (threadIdx.x == 0) {
        d_out[blockIdx.x] = s_a[0];
    }
}

// __global__ void reduce1(float* d_a, float* d_out) {
//     __shared__ float s_a[THREAD_PER_BLOCK];

//     // 搬运数据到共享内存中，每个线程搬运一个元素
//     int global_id = blockDim.x * blockIdx.x + threadIdx.x;
//     s_a[threadIdx.x] = d_a[global_id];
//     // 搬运完需要进行同步
//     __syncthreads();

//     for (int i = blockDim.x / 2; i > 0; i =/2) {
//         if (threadIdx.x < i) {
//             s_a[threadIdx.x] += s_a[threadIdx.x + i];
//         }
//         __syncthreads();
//     }
//     if (threadIdx.x == 0) {
//         d_out[blockIdx.x] = d_a[global_id];
//     }
// }


bool check(float *out,float *res,int n){
    for(int i=0;i<n;i++){
        if (abs(out[i] - res[i]) > 0.005) {
            return false;
        }
    }
    return true;
}

int main() {
    float milliseconds = 0;
    printf("hello \n");

    const int N=32*1024*1024;
    hipSetDevice(0);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);


    float *a=(float *)malloc(N*sizeof(float));
    float *d_a;
    hipMalloc((void **)&d_a,N*sizeof(float));


    int block_num = N / THREAD_PER_BLOCK / 2;

    float *out = (float *)malloc((block_num * sizeof(float)));
    float *d_out;
    hipMalloc((void **)&d_out, block_num * sizeof(float));

    float *res=(float *)malloc((N/THREAD_PER_BLOCK)*sizeof(float));

    for(int i=0;i<N;i++){
        a[i]=1.2f;
    }

    for(int i=0;i<block_num;i++){
        float cur=0;
        for(int j=0;j<THREAD_PER_BLOCK*2;j++){
            cur+=a[i*THREAD_PER_BLOCK*2+j];
        }
        res[i]=cur;
    }

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(block_num);
    dim3 Block(THREAD_PER_BLOCK);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);


    reduce1<<<Grid, Block>>>(d_a, d_out);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);


    hipMemcpy(out, d_out, block_num*sizeof(float),hipMemcpyDeviceToHost);

     if(check(out,res,block_num))printf("the ans is right\n");
    else{
        printf("the ans is wrong\n");
        for(int i=0;i<block_num;i++){
            printf("%lf ",out[i]);
        }
        printf("\n");
    }

    printf("reduce latency = %f ms\n", milliseconds);

    hipFree(d_a);
    hipFree(d_out);

    return 0;
}