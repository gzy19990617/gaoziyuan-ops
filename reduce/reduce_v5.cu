#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


#include <time.h>
#include <sys/time.h>

#define THREAD_PER_BLOCK 256

// v5:展开最后一个warp，减少同步

__global__ void reduce1(float* d_a, float* d_out) {
    volatile __shared__ float s_a[THREAD_PER_BLOCK]; // 不加volatile,会导致精度diff,volatile每次从shaerd取，防止编译器的优化

    int tid = threadIdx.x;
    float* input_begein = d_a + blockIdx.x * blockDim.x * 2;
    s_a[threadIdx.x] = input_begein[threadIdx.x] + input_begein[threadIdx.x + blockDim.x];
    // 搬运完需要进行同步
    __syncthreads();

    for (int i = blockDim.x / 2; i > 32; i /= 2) { // i > 32时才执行
        if (threadIdx.x < i) {
            s_a[threadIdx.x] += s_a[threadIdx.x + i];
            // warp与warp之间没有办法同步，但warp内同线程是可以同步的，所以后面几次迭代不需要同步了
            __syncthreads();
        }
    }
    if (tid < 32) {
        s_a[tid] += s_a[tid + 32];
        s_a[tid] += s_a[tid + 16];
        s_a[tid] += s_a[tid + 8];
        s_a[tid] += s_a[tid + 4];
        s_a[tid] += s_a[tid + 2];
        s_a[tid] += s_a[tid + 1];

    }

    if (threadIdx.x == 0) {
        d_out[blockIdx.x] = s_a[0];
    }
}

// __global__ void reduce1(float* d_a, float* d_out) {
//     __shared__ float s_a[THREAD_PER_BLOCK];

//     // 搬运数据到共享内存中，每个线程搬运一个元素
//     int global_id = blockDim.x * blockIdx.x + threadIdx.x;
//     s_a[threadIdx.x] = d_a[global_id];
//     // 搬运完需要进行同步
//     __syncthreads();

//     for (int i = blockDim.x / 2; i > 0; i =/2) {
//         if (threadIdx.x < i) {
//             s_a[threadIdx.x] += s_a[threadIdx.x + i];
//         }
//         __syncthreads();
//     }
//     if (threadIdx.x == 0) {
//         d_out[blockIdx.x] = d_a[global_id];
//     }
// }


bool check(float *out,float *res,int n){
    for(int i=0;i<n;i++){
        if (abs(out[i] - res[i]) > 0.005) {
            return false;
        }
    }
    return true;
}

int main() {
    float milliseconds = 0;
    printf("hello \n");

    const int N=32*1024*1024;
    hipSetDevice(0);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);


    float *a=(float *)malloc(N*sizeof(float));
    float *d_a;
    hipMalloc((void **)&d_a,N*sizeof(float));


    int block_num = N / THREAD_PER_BLOCK / 2;

    float *out = (float *)malloc((block_num * sizeof(float)));
    float *d_out;
    hipMalloc((void **)&d_out, block_num * sizeof(float));

    float *res=(float *)malloc((N/THREAD_PER_BLOCK)*sizeof(float));

    for(int i=0;i<N;i++){
        a[i]=1.2f;
    }

    for(int i=0;i<block_num;i++){
        float cur=0;
        for(int j=0;j<THREAD_PER_BLOCK*2;j++){
            cur+=a[i*THREAD_PER_BLOCK*2+j];
        }
        res[i]=cur;
    }

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(block_num);
    dim3 Block(THREAD_PER_BLOCK);
    // for (int i = 0; i < 10; i ++) {

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);


        reduce1<<<Grid, Block>>>(d_a, d_out);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);

        printf("reduce latency = %f ms\n", milliseconds);


    // }
    


    hipMemcpy(out, d_out, block_num*sizeof(float),hipMemcpyDeviceToHost);

     if(check(out,res,block_num))printf("the ans is right\n");
    else{
        printf("the ans is wrong\n");
        for(int i=0;i<block_num;i++){
            printf("%lf ",out[i]);
        }
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_out);

    return 0;
}