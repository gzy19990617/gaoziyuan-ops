#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


#include <time.h>
#include <sys/time.h>

#define THREAD_PER_BLOCK 256

// v3:避免banck conflict

// 为了提高内存读写带宽，共享内存被分割成了32个等大小的内存块，即bank。因为一个warp有32个线程，相当于一个线程对应一个内存bank

// bank 0 : 0 32 64 96...
// bank 1: 1 33 65 97...

// 避免同一个warp的线程访问同一个bank，但如果是访问同一个bank中的同一位置，会产生广播，不会发生conflict

__global__ void reduce1(float* d_a, float* d_out) {
    __shared__ float s_a[THREAD_PER_BLOCK];

    // 搬运数据到共享内存中，每个线程搬运一个元素
    float* input_begein = d_a + blockIdx.x * blockDim.x;
    s_a[threadIdx.x] = input_begein[threadIdx.x];
    // 搬运完需要进行同步
    __syncthreads();

    for (int i = 1; i < blockDim.x; i *= 2) {
        if (threadIdx.x < blockDim.x / (2 * i)) {
            // 第一轮：0号线程负责，0+1. 1号线程负责，1+2. 以此类推
            // 第二轮：0号线程负责，0+2. 1号线程负责，4+6. 以此类推
            int index = threadIdx.x * (2 * i);
            s_a[index] += s_a[index + i];
        }
        // 每一次要等这一轮计算完
        __syncthreads();
    }
    //最终每个block把计算结果放在第一个索引的位置
    if (threadIdx.x == 0) {
        d_out[blockIdx.x] = input_begein[0];
    }
}

// __global__ void reduce1(float* d_a, float* d_out) {
//     __shared__ float s_a[THREAD_PER_BLOCK];
    
//     int tid = threadIdx.x;
//     int global_tid = blockIdx.x * blockDim.x + tid;

//     // 搬运数据到共享内存中，每个线程搬运一个元素
//     s_a[tid] = d_a[global_tid];

//     // 搬运完需要进行同步
//     __syncthreads();

//     for (int i = 1; i < blockDim.x; i *= 2) {
//         if (tid % (2 * i) == 0) {
//             d_a[global_tid] += d_a[global_tid + i];
//         }
//         // 每一次要等这一轮计算完
//         __syncthreads();
//     }
//     //最终每个block把计算结果放在第一个索引的位置
//     if (tid == 0) {
//         d_out[blockIdx.x] = d_a[global_tid];
//     }
// }


bool check(float *out,float *res,int n){
    for(int i=0;i<n;i++){
        if (abs(out[i] - res[i]) > 0.005) {
            return false;
        }
    }
    return true;
}

int main() {
    printf("hello \n");

    const int N=32*1024*1024;
    float *a=(float *)malloc(N*sizeof(float));
    float *d_a;
    hipMalloc((void **)&d_a,N*sizeof(float));


    int block_num = N / THREAD_PER_BLOCK;

    float *out = (float *)malloc((block_num * sizeof(float)));
    float *d_out;
    hipMalloc((void **)&d_out, block_num * sizeof(float));

    float *res=(float *)malloc((N/THREAD_PER_BLOCK)*sizeof(float));

    for(int i=0;i<N;i++){
        a[i]=1.2;
    }

    for(int i=0;i<block_num;i++){
        float cur=0;
        for(int j=0;j<THREAD_PER_BLOCK;j++){
            cur+=a[i*THREAD_PER_BLOCK+j];
        }
        res[i]=cur;
    }

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(block_num);
    dim3 Block(THREAD_PER_BLOCK);
    

    reduce1<<<Grid, Block>>>(d_a, d_out);

    hipMemcpy(out, d_out, block_num*sizeof(float),hipMemcpyDeviceToHost);

     if(check(out,res,block_num))printf("the ans is right\n");
    else{
        printf("the ans is wrong\n");
        for(int i=0;i<block_num;i++){
            printf("%lf ",out[i]);
        }
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_out);

    return 0;
}